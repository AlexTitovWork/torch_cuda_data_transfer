#include "hip/hip_runtime.h"
//coded by Alex. 28.10.2021
//alexeytitovwork@gmail.com
//Torch to CUDA data transfer test.

#include <stdio.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include "torch_module_plus_cuda.h"


#include <iostream>
#include <memory>


__global__ void MatrixMulKernel(float *M, float *N, float *P, int width, float *slice /*Torch data*/, int size_slice)
{
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;
    if (Row < width && Col < width)
    {
        float Pvalue = 0;
        for (int i = 0; i < width; ++i)
        {
            Pvalue += M[Row * width + i] * N[width * i + Col];
        }
        P[Row * width + Col] = Pvalue;
    }
    //Reinint data
    if(Row < size_slice)
        slice[Row] = 333.333;
        
}

// void MatMul(float *M, float *N, float *P, int width)
void MatMul(float *M, float *N, float *P, int width, float* data)
{
    float *d_M;
    float *d_N;
    float *d_P;
    float *d_Slice;

    int size = width * width * sizeof(float);

    //allocate CUDA mem for Torch slice
    int size_slice  = 5*sizeof(float); 
    hipMalloc((void **)&d_Slice, size_slice);
    hipMemcpy(d_Slice, data, size_slice, hipMemcpyHostToDevice);

    

    hipMalloc((void **)&d_M, size);
    hipMemcpy(d_M, M, size, hipMemcpyHostToDevice);

    hipMalloc((void **)&d_N, size);
    hipMemcpy(d_N, N, size, hipMemcpyHostToDevice);

    hipMalloc((void **)&d_P, size);

    std :: cout << "Prepared Torch Tensor float * data for loading into CUDA memory:" << std :: endl;
    for(int i = 0; i< 5;i++)
       std::cout << data[i]<< '\n';
      

    dim3 dimGrid(2, 2, 1);
    dim3 dimBlock(width / 2, width / 2, 1);
    // <<<>>> will replace macro KERNEL_ARG2 when compiling 
    MatrixMulKernel <<<dimGrid,dimBlock>>>(d_M, d_M, d_P, width, d_Slice/*Torch data*/, size_slice /*size Td*/);
    hipMemcpy(P, d_P, size, hipMemcpyDeviceToHost);
    //Copy back 
    hipMemcpy(data, d_Slice, size_slice, hipMemcpyDeviceToHost);

    hipFree(d_M);
    hipFree(d_N);
    hipFree(d_P);
}

//classic CUDA call
int test_CUDA(float* data) {

    std::cout << "Start... torch_module_plus_cuda.cu CUDA module \n";

    int elem = 100;
    float *M = new float[elem];
    float *N = new float[elem];
    float *P = new float[elem];

    for (int i = 0; i < elem; ++i)
        M[i] = i;

    for (int i = 0; i < elem; ++i)
        N[i] = i + elem;

    time_t t1 = time(NULL);
  
    MatMul(M, N, P, sqrt(elem), data);
    time_t t2 = time(NULL);
    double seconds = difftime(t2,t1);
    printf ("%.3f seconds total time\n", seconds);
    printf("Calculation result:\n");
    
    for (int i = 0; i < 10; ++i)
        printf("%.3f\t", P[i]);
    printf("\n");
    std::cout << "ok!\n";

    //Reinint data
    for (int i = 0; i < 5; ++i)
        printf("%.3f\t", data[i]);
    
    std::cout << "ok!\n";

   
    delete[] M;
    delete[] N;
    delete[] P;
    return 0;
}